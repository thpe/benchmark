#include "cudachk.h"
#include "comp3d.hpp"
#include "cuda_3d.h"
#include <iostream>

struct Comp3d::impl {
  ~impl ();
  int nx_;
  int ny_;
  int nc_;
  bool allocated;
  struct hipPitchedPtr in;
  struct hipPitchedPtr y_;
  struct hipPitchedPtr coeff;
  struct hipPitchedPtr x;

  struct hipExtent in_extent;
  struct hipExtent y_extent;
  struct hipExtent coeff_extent;
  struct hipExtent x_extent;

  hipMemcpy3DParms in_params;
  hipMemcpy3DParms out_params;

  void alloc (int nc, int nx, int ny);
  void load (uint16_t* data);
  void reduce (int idx);
  void run ();
  void y (float* p);
  void dealloc ();
  void print ();
};

Comp3d::Comp3d() : 
  d_ptr_(std::make_unique<impl>()) 
{
}


Comp3d::Comp3d(const Comp3d& other) :
  d_ptr_(std::make_unique<impl>(*other.d_ptr_)) 
{
}

Comp3d::Comp3d(Comp3d&& other) = default;

Comp3d& Comp3d::operator=(const Comp3d &other) 
{
  *d_ptr_ = *other.d_ptr_;
  return *this;
}

Comp3d& Comp3d::operator=(Comp3d&&) = default;

Comp3d::~Comp3d() = default;


void
Comp3d::run ()
{
  d_ptr_->run ();
}

void
Comp3d::y (float* p)
{
  std::cout << std::hex << p << std::dec << std::endl; 
  d_ptr_->y (p);
}

void
Comp3d::alloc (int nc, int nx, int ny)
{
  d_ptr_->alloc (nc, nx, ny);
}

void
Comp3d::load (uint16_t* data)
{
  d_ptr_->load (data);
}

void
Comp3d::print ()
{
  d_ptr_->print ();
}

void
Comp3d::impl::alloc (int nc, int nx, int ny)
{
  std::cout << "alloc...";
  x_extent = make_hipExtent(nc*sizeof(float), nx, ny);
  in_extent = make_hipExtent(1*sizeof(uint16_t), nx * 16, ny * 16);
  y_extent = make_hipExtent(nx*sizeof(float), ny, 1);
  coeff_extent = make_hipExtent(nc*sizeof(float), 1, 1);
  cudaChk(hipMalloc3D (&x,     x_extent));
  cudaChk(hipMalloc3D (&in,    in_extent));
  cudaChk(hipMalloc3D (&y_,    y_extent));
  cudaChk(hipMalloc3D (&coeff, coeff_extent));
  cudaChk(hipMemset3D (x,     0.0f, x_extent));
  cudaChk(hipMemset3D (in,    0u,   in_extent));
  cudaChk(hipMemset3D (y_,    0.0f, y_extent));
  cudaChk(hipMemset3D (coeff, 0.0f, coeff_extent));
  allocated = true;

  in_params        = {0};
  in_params.kind   = hipMemcpyHostToDevice;
  in_params.dstPtr = in;
  in_params.srcPos = make_hipPos(0,0,0);
  in_params.dstPos = make_hipPos(0,0,0);
  in_params.extent = in_extent;

  out_params        = {0};
  out_params.kind   = hipMemcpyDeviceToHost;
  out_params.srcPtr = y_;
  out_params.srcPos = make_hipPos(0,0,0);
  out_params.dstPos = make_hipPos(0,0,0);
  out_params.extent = y_extent;

  nx_ = nx;
  ny_ = ny;
  nc_ = nc;
  
  std::cout << "done.\n";
}

void
Comp3d::impl::print ()
{
  print_int<<< 1, 1 >>> (480, 640, 1, in);
  print_int<<< 1, 1 >>> (30, 40, 1, y_);
}

void
Comp3d::impl::load (uint16_t* data)
{
  in_params.srcPtr = make_hipPitchedPtr(data, 2, 1, 640);
  std::cout << "val " << data[0] << "\n";
  cudaChk(hipMemcpy3D (&in_params));
}

void
Comp3d::impl::run (void)
{
    dim3 gdim (30, 40, 1);
    dim3 bdim (1, 1, 1);
    in_reduce<<< gdim, bdim >>> (1, 0, (uint16_t*)in.ptr, (float*)y_.ptr);
}


std::ostream& operator<<(std::ostream& os, const hipExtent& p)
{
    os << p.width << ", " << p.height << ", " << p.depth;
    return os;
}
std::ostream& operator<<(std::ostream& os, const hipPos& p)
{
    os << p.x << ", " << p.y << ", " << p.z;
    return os;
}
std::ostream& operator<<(std::ostream& os, const hipPitchedPtr& p)
{
    os << "ptr " << std::hex << p.ptr << std::dec << " pitch: " << p.pitch << ", " << p.xsize << " x " << p.ysize;
    return os;
}
std::ostream& operator<<(std::ostream& os, const hipMemcpy3DParms& p)
{
    os << p.srcPtr<< " (" << p.srcPos<< ")" <<" -> " << p.dstPtr<< " (" << p.srcPos<< ")" << " ext " << p.extent;
    return os;
}

void
Comp3d::impl::y (float* y)
{
  out_params.dstPtr = make_hipPitchedPtr(y, 512, nx_ * sizeof(float), ny_);
  std::cout << out_params << std::endl;
  cudaChk(hipMemcpy3D (&out_params));
}

void
Comp3d::impl::dealloc ()
{
  std::cout << "dealloc...";
  hipFree(x.ptr);
  hipFree(in.ptr);
  hipFree(y_.ptr);
  hipFree(coeff.ptr);
  allocated = false;
  std::cout << "done.\n";
}

Comp3d::impl::~impl ()
{
  dealloc ();
}
